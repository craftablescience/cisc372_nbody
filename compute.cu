#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#include "config.h"
#include "vector.h"

__global__ void computePairwiseAcceleration(vector3* hPos, vector3* values, vector3** accels, double* mass) {
	// Row
	int i = threadIdx.x;
	if (i >= NUMENTITIES) {
		return;
	}

	accels[i] = &values[i * NUMENTITIES];

	for (int j = 0; j < NUMENTITIES; j++) {
		if (i == j) {
			FILL_VECTOR(accels[i][j], 0, 0, 0);
		} else {
			vector3 dist;
			for (int k = 0; k < 3; k++) {
				dist[k] = hPos[i][k] - hPos[j][k];
			}
			double mag_sq = dist[0] * dist[0] + dist[1] * dist[1] + dist[2] * dist[2];
			double mag = sqrt(mag_sq);
			double accel = -GRAV_CONSTANT * mass[j] / mag_sq;
			FILL_VECTOR(accels[i][j], accel * dist[0] / mag, accel * dist[1] / mag, accel * dist[2] / mag);
		}
	}
}

__global__ void computeSum(vector3* hVel, vector3* hPos, vector3** accels) {
	// Row
	int i = threadIdx.x;
	if (i >= NUMENTITIES) {
		return;
	}

	vector3 sum{0, 0, 0};
	for (int j = 0; j < NUMENTITIES; j++) {
		for (int k = 0; k < 3; k++) {
			sum[k] += accels[i][j][k];
		}
	}
	for (int k = 0; k < 3; k++) {
		hVel[i][k] += sum[k] * INTERVAL;
		hPos[i][k] += hVel[i][k] * INTERVAL;
	}
}

extern vector3* hVel;
extern vector3* hPos;
extern double*  mass;

//compute: Updates the positions and locations of the objects in the system based on gravity
//Parameters: None
//Returns: None
//Side Effect: Modifies the hVel and hPos arrays with the new positions and accelerations
//             after 1 INTERVAL
void compute() {
	vector3*  dhVel;
	vector3*  dhPos;
	vector3*  values;
	vector3** accels;
	double*   dMass;

	hipMalloc(&dhVel,  sizeof(vector3) * NUMENTITIES);
	hipMalloc(&dhPos,  sizeof(vector3) * NUMENTITIES);
	hipMalloc(&values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	hipMalloc(&accels, sizeof(vector3) * NUMENTITIES);
	hipMalloc(&dMass,  sizeof(double)  * NUMENTITIES);

	hipMemcpy(dhVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(dhPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(dMass, mass, sizeof(double)  * NUMENTITIES, hipMemcpyHostToDevice);

	computePairwiseAcceleration<<<1,108>>>(dhPos, values, accels, dMass);
	hipDeviceSynchronize();

	computeSum<<<1,108>>>(dhVel, dhPos, accels);
	hipDeviceSynchronize();

	hipMemcpy(hPos, dhPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hVel, dhVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);

	hipFree(dMass);
	hipFree(accels);
	hipFree(dhPos);
	hipFree(dhVel);
}
